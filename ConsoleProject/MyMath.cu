#include "hip/hip_runtime.h"
#include "pch.h"
#include "MyMath.h"

__host__ __device__
float MyMath::Dot(const MyMath::Vector3& v1, const MyMath::Vector3& v2)
{
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__host__ __device__
float MyMath::Dot(const MyMath::Vector4& v1, const MyMath::Vector4& v2)
{
	return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z + v1.w * v2.w;
}

__host__ __device__
MyMath::Vector3 MyMath::Cross(const MyMath::Vector3& v1, const MyMath::Vector3& v2)
{
	return MyMath::Vector3(v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x);
}

//Generates good assembly this way.
__host__ __device__
float MyMath::Clamp(const float val, const float min, const float max)
{
	const float result = val < min ? min : val;
	return result > max ? max : result;
}

__host__ __device__
bool MyMath::FloatEquals(float f1, float f2)
{
	return abs(f1 - f2) < FLT_EPSILON;
}
