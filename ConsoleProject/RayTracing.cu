#include "hip/hip_runtime.h"
﻿#include "pch.h"
#include "RayTracing.h"

#include "ANSIRGB.h"

__global__ void UpdateObjects(
	Object3D** objects,
	unsigned int count,
	double dt
)
{
	size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= count)
	{
		return;
	}
	//Do culling, oct-tree, occlusion
	//When this is implemented we have to make it its own kernel since all objects oct-tree node has to be updated before physics update starts.

	//Do physics against objects in oct-tree nodes next to this object's node
	Object3D* object = objects[index];
	switch (object->GetType())
	{
	case ObjectType::SphereType:
	{
		((Sphere*)object)->Update(dt);
		break;
	}
	case ObjectType::PlaneType:
	{
		((Plane*)object)->Update(dt);
		break;
	}
	default:
	{
		break;
	}
	}
	return;
}

__global__ void Culling(

)
{

}

__global__ void RayTrace(
	Object3D* DEVICE_MEMORY_PTR const objects,
	const unsigned int count,
	const RayTracingParameters* params,
	char* resultArray,
	const PrintMachine::PrintMode mode
)
{
	const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t column = blockIdx.x * blockDim.x + threadIdx.x;
	
	//Localization of variables.
	const size_t x = params->x;
	const size_t y = params->y;

	if (column >= x || row >= y)
	{
		return;
	}

	//Set the amount of characters in the buffer per pixel depending on the mode.
	const size_t size = (mode == PrintMachine::ASCII || mode == PrintMachine::PIXEL) ? 12 : 20;

	//#todo: Do this when minimizing instead of in a thread.
	//If the pixel is at the end of a line, output \n and return.
	if (column == (x - 1))
	{
		resultArray[row * (x * size) + column * size] = '\n';
		return;
	}

	//Convert pixel coordinates to (clip space? screen space?)
	const float convertedY = ((float)y - row * 2) / y;
	const float convertedX = (2 * column - (float)x) / x;

	//Localization of variables.
	MyMath::Vector3 cameraPos = params->camPos;
	const float element1 = params->element1;
	const float element2 = params->element2;
	const MyMath::Matrix inverseVMatrix = params->inverseVMatrix;
	const float camFarDist = params->camFarDist;

	//Calculate the ray.
	const MyMath::Vector4 pixelVSpace = MyMath::Vector4(convertedX * element1, convertedY * element2, 1.0f, 0.0f);
	const MyMath::Vector3 directionWSpace = inverseVMatrix.Mult(pixelVSpace).xyz().Normalize_InPlace();
	
	//Used during intersection tests with spheres.
	const float a = Dot(directionWSpace, directionWSpace);
	const float fourA = 4.0f * a;
	const float divTwoA = 1.0f / (2.0f * a);

	char data = ' ';
	float closest = 99999999.f;
	float shadingValue = 0.0f;
	MyMath::Vector3 bestColor;
	MyMath::Vector3 bestNormal;
	
	//Ray trace against every object.
	for (size_t i = 0; i < count; i++)
	{
		//#todo: Here we need to check if the object is culled, if it is we continue on the next object.
		
		const ObjectType type = objects[i]->GetType();

		//Ray-Sphere intersection test.
		if (type == ObjectType::SphereType)
		{
			const Sphere* sphere = (Sphere*)objects[i];
			const MyMath::Vector3 spherePos = sphere->GetPos();

			const MyMath::Vector3 objectToCam = cameraPos - spherePos;
			const float radius = sphere->GetRadius();

			const float b = 2.0f * Dot(directionWSpace, objectToCam);
			const float c = Dot(objectToCam, objectToCam) - (radius * radius);

			const float discriminant = b * b - fourA * c;

			//It hit
			if (discriminant >= 0.0f)
			{
				const float sqrtDiscriminant = sqrt(discriminant);
				const float minusB = -b;
				float t1 = (minusB + sqrtDiscriminant) * divTwoA;
				const float t2 = (minusB - sqrtDiscriminant) * divTwoA;

				//Remove second condition to enable "backface" culling for spheres. IE; not hit when inside them.
				if (t1 > t2 && t2 >= 0.0f)
				{
					t1 = t2;
				}

				if (t1 < closest && t1 > 0.0f)
				{
					closest = t1;
					const MyMath::Vector3 normalSphere = (cameraPos + directionWSpace * closest - spherePos).Normalize();
					bestNormal = normalSphere;

					//1, 0, 0 is just temporary light direction.
					//#todo: INTRODUCE REAL LIGHTS!
					shadingValue = Dot(normalSphere, MyMath::Vector3(1.0f, 0.0f, 0.0f));
					bestColor = sphere->GetColor();
				}
			}
		}
		else if (type == ObjectType::PlaneType)
		{
			const Plane* plane = (Plane*)objects[i];
			const MyMath::Vector3 planeNormal = plane->GetNormal();
			const MyMath::Vector3 planePos = plane->GetPos();

			const float dotLineAndPlaneNormal = Dot(directionWSpace, planeNormal);

			//Check if the line and plane are paralell, if not it hit.
			if (!MyMath::FloatEquals(dotLineAndPlaneNormal, 0.0f))
			{
				float t1 = Dot((planePos - cameraPos), planeNormal) / dotLineAndPlaneNormal;

				if (t1 > 0.0f)
				{
					if (t1 < closest)
					{
						MyMath::Vector3 point = cameraPos + (directionWSpace * t1);
						const float halfPlaneWidth = plane->GetWidth() * 0.5f;
						const float halfPlaneHeight = plane->GetHeight() * 0.5f;

						//If the ray hit inbetween the width & height.
						if (
							point.x > planePos.x - halfPlaneWidth && point.x < planePos.x + halfPlaneWidth &&	//Width
							point.z > planePos.z - halfPlaneHeight && point.z < planePos.z + halfPlaneHeight	//Height
						)
						{
							//1, 0, 0 is just temporary light direction.
							//#todo: INTRODUCE REAL LIGHTS!
							shadingValue = Dot(planeNormal, MyMath::Vector3(1.0f, 0.0f, 0.0f));

							//Comment in this if statement to get "backface" culling for planes.
							//if (shadingValue > 0.0f) {
							closest = t1;
							//}

							bestColor = plane->GetColor();
							bestNormal = planeNormal;

							//Reverse the normal if viewed from backside.
							if (dotLineAndPlaneNormal > 0.0f)
							{
								bestNormal *= -1;
							}
						}
					}
				}
			}
			
		}
	}

	//Decide what character to write for this pixel.
	static const float t = 67;
	//If we miss or its outside the frustum we dont print anything.
	if (closest > camFarDist)
	{
		data = ascii[0];
	}
	else
	{
		const int dataIndex = MyMath::Clamp((int)ceil(shadingValue * t), 1, NUM_ASCII_CHARACTERS); //Clamp with 1 as min, so that the empty space does not get used.
		data = ascii[dataIndex];
	}
	
	//Now we need to take the raytraced information and output it to our result array of chars.
	//If the mode is not RGB we need to convert the colors to 8bit.
	if (mode == PrintMachine::PIXEL || mode == PrintMachine::ASCII)
	{
		//If the pixel hit something during ray tracing.
		if (data != ' ')
		{
			float ambient = 0.01492537f * 19;
			if (shadingValue < ambient)
			{
				shadingValue = ambient;
			}
			//Apply shading.
			bestColor *= shadingValue;

			//Convert the 24bit RGB color to ANSI 8 bit color.
			uint8_t index = ansi256_from_rgb(((uint8_t)bestColor.x << 16) + ((uint8_t)bestColor.y << 8) + (uint8_t)bestColor.z);
			uint8_t originalIndex = index;
			//Now we need to convert this number (0-255) to 3 chars.
			uint8_t tens = index % 100;
			uint8_t singles = tens % 10;
			char first = '\0';
			char second = '\0';
			char third = '\0';

			if (index >= 100)
			{
				index = (uint8_t)((index - tens) * 0.01f);
				first = index + '0';
			}
			if (tens >= 10 || originalIndex >= 100)
			{
				tens = (uint8_t)((tens - singles) * 0.1f);
				second = tens + '0';
			}
			third = singles + '0';

			//If in ASCII mode we change foreground color and also print the value in data.
			if (mode == PrintMachine::ASCII)
			{
				char finalData[12] = {
					'\x1b', '[',			//Escape character
					'3', '8', ';',			//Keycode for foreground
					'5', ';',				//Keycode for foreground
					first, second, third,	//Index
					'm', data				//Character data.
				};
				memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char) * size);
			}
			//If in PIXEL mode we change background color and do not print the value.
			else //If in pixel mode we only print the color.
			{
				char finalData[12] = {
					'\x1b', '[',			//Escape character
					'4', '8', ';',			//Keycode for background
					'5', ';',				//Keycode for background
					first, second, third,	//Index
					'm', ' '				//Character data.
				};
				memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char) * size);
			}
		}
		//If it is an empty space we can not use a background color.
		else
		{
			char finalData[12] = {
				'\x1b', '[',			//Escape character
				'4', '8', ';',			//Keycode for background
				'5', ';',				//Keycode for background
				'\0', '1', '6',			//Index
				'm', ' '				//Character data.
			};
			memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char) * size);
		}
	}
	//If the mode is in any of the RGB modes we simply use the rgb values gathered.
	else
	{
		//If the pixel hit something during ray tracing.
		if (data != ' ')
		{
			//Increase the right-hand value to increase the ambient light.
			float ambient = 0.01492537f * 7;
			if (shadingValue < ambient)
			{
				shadingValue = ambient;
			}
			//Apply shading.
			bestColor *= shadingValue;

			//Needed to print the rgb values to final data.
			char firstR = '\0';
			char secondR = '\0';
			char thirdR = '\0';

			char firstG = '\0';
			char secondG = '\0';
			char thirdG = '\0';

			char firstB = '\0';
			char secondB = '\0';
			char thirdB = '\0';

			//R
			uint8_t originalIndex;
			uint8_t index;
			if (mode == PrintMachine::RGB_NORMALS)
			{
				originalIndex = (uint8_t)(bestNormal.x * 255);
				index = (uint8_t)(bestNormal.x * 255);
			}
			else
			{
				originalIndex = (uint8_t)bestColor.x;
				index = (uint8_t)bestColor.x;
			}
			
			uint8_t tens = index % 100;
			uint8_t singles = tens % 10;

			if (index >= 100)
			{
				index = (uint8_t)((index - tens) * 0.01f);
				firstR = index + '0';
			}
			if (tens >= 10 || originalIndex >= 100)
			{
				tens = (uint8_t)((tens - singles) * 0.1f);
				secondR = tens + '0';
			}
			thirdR = singles + '0';

			//G
			if (mode == PrintMachine::RGB_NORMALS)
			{
				originalIndex = (uint8_t)(bestNormal.y * 255);
				index = (uint8_t)(bestNormal.y * 255);
			}
			else
			{
				originalIndex = (uint8_t)bestColor.y;
				index = (uint8_t)bestColor.y;
			}

			tens = index % 100;
			singles = tens % 10;

			if (index >= 100)
			{
				index = (uint8_t)((index - tens) * 0.01f);
				firstG = index + '0';
			}
			if (tens >= 10 || originalIndex >= 100)
			{
				tens = (uint8_t)((tens - singles) * 0.1f);
				secondG = tens + '0';
			}
			thirdG = singles + '0';

			//B
			if (mode == PrintMachine::RGB_NORMALS)
			{
				originalIndex = (uint8_t)(bestNormal.z * 255);
				index = (uint8_t)(bestNormal.z * 255);
			}
			else
			{
				originalIndex = (uint8_t)bestColor.z;
				index = (uint8_t)bestColor.z;
			}

			tens = index % 100;
			singles = tens % 10;

			if (index >= 100)
			{
				index = (uint8_t)((index - tens) * 0.01f);
				firstB = index + '0';
			}
			if (tens >= 10 || originalIndex >= 100)
			{
				tens = (uint8_t)((tens - singles) * 0.1f);
				secondB = tens + '0';
			}
			thirdB = singles + '0';

			//If in ASCII mode we change foreground color and also print the value in data.
			if (mode == PrintMachine::RGB_ASCII)
			{
				char finalData[20] = {
					'\x1b', '[',					//Escape character
					'3', '8', ';',					//Keycode for foreground
					'2', ';',						//Keycode for foreground
					firstR, secondR, thirdR, ';',	//R
					firstG, secondG, thirdG, ';',	//G
					firstB, secondB, thirdB,		//B
					'm', data						//Character data.
				};
				memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char)* size);
			}
			else if (mode == PrintMachine::RGB_PIXEL)
			{
				char finalData[20] = {
					'\x1b', '[',					//Escape character
					'4', '8', ';',					//Keycode for foreground
					'2', ';',						//Keycode for foreground
					firstR, secondR, thirdR, ';',	//R
					firstG, secondG, thirdG, ';',	//G
					firstB, secondB, thirdB,		//B
					'm', ' '						//Character data.
				};
				memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char)* size);
			}
			//Normals.
			else
			{
				char finalData[20] = {
					'\x1b', '[',					//Escape character
					'4', '8', ';',					//Keycode for foreground
					'2', ';',						//Keycode for foreground
					firstR, secondR, thirdR, ';',	//R
					firstG, secondG, thirdG, ';',	//G
					firstB, secondB, thirdB,		//B
					'm', ' '						//Character data.
				};
				memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char) * size);
			}
		}
		//If it is an empty space we can not use a background color. 
		else
		{
			char finalData[20] = {
				'\x1b', '[',			//Escape character
				'4', '8', ';',			//Keycode for background
				'2', ';',				//Keycode for background
				'\0', '\0', '0', ';',	//R
				'\0', '\0', '0', ';',	//G
				'\0', '\0', '0',		//B
				'm', ' '				//Character data.
			};
			memcpy(resultArray + (row * (x * size) + column * size), finalData, sizeof(char)* size);
		}
	}
	
	return;
}

RayTracer::RayTracer()
{
	const size_t size = PrintMachine::GetMaxSize();

	hipMalloc(&m_deviceResultArray, sizeof(char) * size);

	//Allocate the array which will contain the full screen before minimization.
	m_hostResultArray = std::make_unique<char[]>(size);

	//Allocate the minimized array which will be printed to the console.
	m_minimizedResultArray = std::make_unique<char[]>(size);
}

RayTracer::~RayTracer()
{
	hipFree(m_deviceResultArray);
}

void RayTracer::RayTracingWrapper(
	const size_t x,
	const size_t y,
	const DeviceObjectArray<Object3D*>& deviceObjects,
	const RayTracingParameters DEVICE_MEMORY_PTR rayTracingParameters,
	double dt
)
{
	//The backbuffer needs to be reset in order to not produce artefacts, especially when switching printing mode to RGB.
	ResetDeviceBackBuffer();

	//Update the objects. 1 thread per object.
	unsigned int threadsPerBlock = deviceObjects.count;
	unsigned int numberOfBlocks = 1;
	if (deviceObjects.count > 1024)
	{
		numberOfBlocks = static_cast<int>(std::ceil(deviceObjects.count / 1024.0));
	}
	dim3 gridDims(numberOfBlocks, 1, 1);
	dim3 blockDims(threadsPerBlock, 1, 1);
	
	//If it is the first rendering loop we need to construct the octtree, so that we can access it in the physicsupdate. But otherwise it only has to be done after the physics update.
	// 
	//Physics update of the objects.
	UpdateObjects<<<gridDims, blockDims>>>(
		deviceObjects.m_deviceArray,
		deviceObjects.count,
		dt
	);

	//Classify the objects into the octtree.
	//Mark objects within the frustum
	/*
	Culling<<<gridDims, blockDims>>>(
		deviceObjects.using1st ? deviceObjects.m_deviceArray1 : deviceObjects.m_deviceArray2,

	);
	*/
	//After we do the culling we check the remaining objects within the octtree and update their closest position to the camera.


	//Do the raytracing. Calculate x and y dimensions in blocks depending on screensize.
	//1 thread per pixel.
	gridDims.x = static_cast<unsigned int>(std::ceil((x + 1) / 16.0));
	gridDims.y = static_cast<unsigned int>(std::ceil(y / 16.0));
	blockDims.x = 16u;
	blockDims.y = 16u;
	
	RayTrace<<<gridDims, blockDims>>>(
		deviceObjects.m_deviceArray,
		deviceObjects.count,
		rayTracingParameters,
		m_deviceResultArray,
		PrintMachine::GetPrintMode()
	);
	//Make sure all the threads are done with the ray tracing.
	gpuErrchk(hipDeviceSynchronize());

	//#todo: Make a function to get the original "max size". Why?
	const size_t size = PrintMachine::GetMaxSize();

	//Copy all data from GPU -> CPU.
	gpuErrchk(hipMemcpy(m_hostResultArray.get(), m_deviceResultArray, size, hipMemcpyDeviceToHost));

	//Minimize the result, by removing unneccessary ANSI escape sequences.
	size_t newSize = MinimizeResults(size, y);

	//Locking/unlocking of the mutex, flagging for changing buffer, and changing the printing size now all happens within this function.
	//-----------------------------------------------------------------------------------------------------
	PrintMachine::SetDataInBackBuffer(m_minimizedResultArray.get(), newSize);
	//-----------------------------------------------------------------------------------------------------

	return;
}

void RayTracer::ResetDeviceBackBuffer()
{
	const size_t size = PrintMachine::GetMaxSize();
	hipMemset(m_deviceResultArray, 0, size);
}

size_t RayTracer::MinimizeResults(const size_t size, const size_t y)
{
	PrintMachine::PrintMode mode = PrintMachine::GetPrintMode();


	//If its in 8 bit mode.
	if (mode == PrintMachine::ASCII || mode == PrintMachine::PIXEL)
	{
		return Minimize8bit(size, y);
	}
	//Else it is rgb.
	else
	{
		return MinimizeRGB(size, y);
	}
}

size_t RayTracer::Minimize8bit(const size_t size, const size_t y)
{
	size_t newlines = 0;
	size_t addedChars = 0;

	//We hold a pointer to the spot in the buffer with the latest color.
	char* latestColor = nullptr;

	for (size_t i = 0; i < size;)
	{
		char current = m_hostResultArray[i];

		//If we are handling a pixel.
		if (current == '\x1b')
		{
			//If its not the same color add the whole escape sequence and update latest color.
			if (
				!latestColor ||
				latestColor[0] != m_hostResultArray[i + 7] ||
				latestColor[1] != m_hostResultArray[i + 8] ||
				latestColor[2] != m_hostResultArray[i + 9]
			)
			{
				//Move the pointer to the spot in the array with the color.
				latestColor = m_hostResultArray.get() + i + 7;

				//Copy the escape sequence and data to the minimized result.
				memcpy(m_minimizedResultArray.get() + addedChars, m_hostResultArray.get() + i, 12);

				addedChars += 12;
			}
			//Only add the data and not the escape sequence.
			else
			{
				m_minimizedResultArray[addedChars] = m_hostResultArray[i + 11];

				addedChars += 1;
			}

			//Move 12 characters forward.
			i += 12;
		}
		//If we are handling the end of a line.
		else if (current == '\n')
		{
			++newlines;

			m_minimizedResultArray[addedChars] = '\n';
			++addedChars;

			//Move 1 character forward.
			++i;

			//Stop iterating if the amount of lines equals the height.
			if (newlines == y)
			{
				break;
			}
		}
		//For \0. Simply move one character forward.
		else
		{
			++i;
		}
	}

	return addedChars;
}

size_t RayTracer::MinimizeRGB(const size_t size, const size_t y)
{
	size_t newlines = 0;
	size_t addedChars = 0;

	//We hold a pointer to the spot in the buffer with the latest color.
	char* latestColor = nullptr;

	for (size_t i = 0; i < size;)
	{
		char current = m_hostResultArray[i];

		//If we are handling a pixel.
		if (current == '\x1b')
		{
			//If its not the same color Add the escape sequence and update latest color.
			if (
				!latestColor ||
				latestColor[0] != m_hostResultArray[i + 7] || latestColor[1] != m_hostResultArray[i + 8] || latestColor[2] != m_hostResultArray[i + 9] ||		//R
				latestColor[4] != m_hostResultArray[i + 11] || latestColor[5] != m_hostResultArray[i + 12] || latestColor[6] != m_hostResultArray[i + 13] ||	//G
				latestColor[8] != m_hostResultArray[i + 15] || latestColor[9] != m_hostResultArray[i + 16] || latestColor[10] != m_hostResultArray[i + 17]		//B
			)
			{
				//Move the pointer to the spot in the array with the color.
				latestColor = m_hostResultArray.get() + i + 7;

				//Copy the escape sequence and data to the minimized result.
				memcpy(m_minimizedResultArray.get() + addedChars, m_hostResultArray.get() + i, 20);

				addedChars += 20;
			}
			//Only add the data and not the escape sequence.
			else
			{
				m_minimizedResultArray[addedChars] = m_hostResultArray[i + 19];

				addedChars += 1;
			}

			//Move 20 characters forward.
			i += 20;
		}
		//If we are handling the end of a line.
		else if (current == '\n')
		{
			++newlines;

			m_minimizedResultArray[addedChars] = m_hostResultArray[i];
			++addedChars;

			//Move 1 character forward.
			++i;

			//Stop iterating if the amount of lines equals the height.
			if (newlines == PrintMachine::GetHeight())
			{
				break;
			}
		}
		//For \0. Simply move 1 character forward.
		else
		{
			++i;
		}
	}

	return addedChars;
}
